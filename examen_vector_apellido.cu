#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

/*
Apellido paterno: GARCIA
 La primer letra del apellido es: G (corresponde a la septima letra del abecedario)
 entonces 
 tam_vector= 7*500= 3500 elementos

 La ultima letra es: A (corresponde a la primer letra del abecedario)
 entonces 
 num_hilos_por_bloque= 1*4= 4

 

tambloque= 4 por lo anteriormente calculado y restriccion del problema seran 4 hilos solamente por bloque
entonces
numbloques= (N+tambloque-1) / tambloque    -->   donde tambloque=4 y N= 3500 
entonces
numbloques= (3500+4-1) / 4 = 875
*/

// Nota: Se hicieron pruebas por ejemplo con tam_vector de 10 (elementos) y se noto que si bien hacia el calculo, imprimia de forma incorrecta, asi que se determino que el programa solo funciona con el tamaño de vectores,numero de bloque y tamaño de bloque previamente calculados


__global__ void MultiplicarVectores (int * vectorA_dispositivo, int * vectorB_dispositivo, int * vectorC_dispositivo, int N) // N es el tam_vector
{
    int idHilo= (blockIdx.x * blockDim.x) + threadIdx.x; 
    if (idHilo < N)
    {
        vectorC_dispositivo[idHilo]= vectorA_dispositivo[idHilo] * vectorB_dispositivo[idHilo]; // cada hilo recibe un id de la posicion que va a trabajar
    }
}




int main(int argc, char const *argv[])
{
    int tam_vector= 30; 
    // int tam_vector= 10; // tamaño de 10 (para probar)

    srand(time(NULL)); // para utilizar valores aleatorios 

    // Declaracion de vectores
    int *vectorA_dispositivo, *vectorB_dispositivo, *vectorC_dispositivo;
    int *vectorA_host, *vectorB_host, *vectorC_host;


    // Manejo de los arreglos en el host
    vectorA_host= (int *)malloc(tam_vector*sizeof(int));
    vectorB_host= (int *)malloc(tam_vector*sizeof(int));
    vectorC_host= (int *)malloc(tam_vector*sizeof(int));


    // inicializar arreglos del host con valores aleatorios
    for (int i=0; i<tam_vector; i++)
    {
        //vectorA_host[i]= 10 + rand() % 11; // valores aleatorios entre 0 y 10
        vectorA_host[i]= i+1; // valores sucesivos a partir de 1 (para probar)
    }
    for (int i=0; i<tam_vector; i++)
    {
        //vectorB_host[i]= 10 + rand() % 11; // valores aleatorios entre 0 y 10
        vectorB_host[i]= i+1; // valores sucesivos a partir de 1 (para probar)
    }
    for (int i=0; i<tam_vector; i++)
    {
        //vectorC_host[i]= 10 + rand() % 11; // valores aleatorios entre 0 y 10
        vectorC_host[i]= i+1; // valores sucesivos a partir de 1 (para probar)
    }


    // Declaracion de la memoria en el dispositivo (GPU)
    hipMalloc(&vectorA_dispositivo, tam_vector*sizeof(int));
    hipMalloc(&vectorB_dispositivo, tam_vector*sizeof(int));
    hipMalloc(&vectorC_dispositivo, tam_vector*sizeof(int));


    // Copiar del host al dispositivo
    hipMemcpy(vectorA_dispositivo, vectorA_host, tam_vector*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(vectorB_dispositivo, vectorB_host, tam_vector*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(vectorC_dispositivo, vectorC_host, tam_vector*sizeof(int),hipMemcpyHostToDevice);
    
  
    // Tamaño del bloque y numero de bloques obtenidos del calculo previo (se encuentra debajo de las bibliotecas)
    int tamanio_bloque= 4;
    int num_bloques= 875; 
    

    // Medir tiempo del dispositivo (GPU) usando eventos de cuda
    hipEvent_t inicio, fin; // vendria siendo el equivalente a clock_t 
    // se declaran variables que van a ser eventos
    hipEventCreate(&inicio);
    hipEventCreate(&fin);



    hipEventRecord(inicio); // se marca en donde va a empezar a medir el tiempo de GPU que es cuando se lanza el kernel para empezar a hacer calculos
    // Lanzar el kernel
    MultiplicarVectores<<<num_bloques,tamanio_bloque>>>(vectorA_dispositivo, vectorB_dispositivo, vectorC_dispositivo, tam_vector); // kernel que va a trabajar
    hipEventRecord(fin); // se marca en donde va a terminar de medirse el tiempo de GPU que es cuando el kernel ya acabo


    hipDeviceSynchronize(); // es el equivalente a .join en hilos. Es importante utilizarlo para no trabar la GPU


    // Copiar del dispositivo al host
    hipMemcpy(vectorC_host, vectorC_dispositivo, tam_vector*sizeof(int),hipMemcpyDeviceToHost);



    printf("\nLos primeros 5 elementos son:\n");
    for (int i=0; i<5; i++)
    {
        printf("%d ", vectorC_host[i]);
    }
    printf("\n");



    printf("\nLos ultimos 5 elementos son:\n");
    for (int i=tam_vector-5; i<tam_vector; i++)
    {
        printf("%d ", vectorC_host[i]);
    }
    printf("\n");


    // Calcular el tiempo que le tomo a la GPU hacer los calculos
    float tiempo_GPU=0; 
    hipEventElapsedTime(&tiempo_GPU, inicio, fin); // se indica en donde se va a guardar, el inicio y el final. Siempre va a devolver el tiempo en milisegundos
    printf("\nEl tiempo de ejecucion del dispositivo (GPU) fue de: %f segundos\n",tiempo_GPU/1000); // se divide tiempo/1000 para dar el tiempo en segundos en vez de milisegundos



    // Liberar memoria del dispositivo (GPU)
    hipFree(vectorA_dispositivo);
    hipFree(vectorB_dispositivo);
    hipFree(vectorC_dispositivo);
    

    // Liberar memoria del host (CPU)
    free(vectorA_host);
    free(vectorB_host);
    free(vectorC_host);
   

    return 0;
}
