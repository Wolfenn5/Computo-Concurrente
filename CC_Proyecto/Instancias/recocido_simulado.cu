#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#define MAX_CITIES 1000

typedef struct {
    double x[MAX_CITIES];
    double y[MAX_CITIES];
    int num_cities;
} TSPData;

__device__ double distance(double x1, double y1, double x2, double y2){
    return sqrt((x1 - x2)*(x1 - x2) + (y1 - y2)*(y1 - y2));
}

__device__ double total_distance(TSPData *data, int *tour){
    double dist = 0.0;
    for(int i = 0; i < data->num_cities - 1; i++){
        dist += distance(data->x[tour[i]], data->y[tour[i]], data->x[tour[i+1]], data->y[tour[i+1]]);
    }
    dist += distance(data->x[tour[data->num_cities-1]], data->y[tour[data->num_cities-1]], data->x[tour[0]], data->y[tour[0]]);
    return dist;
}

__device__ void swap(int *a, int *b){
    int tmp = *a;
    *a = *b;
    *b = tmp;
}

__device__ void shuffle(int *tour, int n, hiprandState *state) {
    for(int i = 0; i < n; i++) tour[i] = i;
    for(int i = 0; i < n; i++) {
        int j = hiprand(state) % n;
        int tmp = tour[i];
        tour[i] = tour[j];
        tour[j] = tmp;
    }
}




__global__ void kernel_simulated_annealing(TSPData *data, int *best_tours, double *best_costs, int max_iter, unsigned long seed) 
{
    int idHilo = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Inicializar el estado del generador curand (para aleatoiros)
    hiprandState state;
    hiprand_init(seed, idHilo, 0, &state); // semilla, idHilo es el id unico de hilo


    // Cada hilo tendrá su propio tour y estado (semilla para aleatorios)
    int current_tour[MAX_CITIES];
    shuffle(current_tour, data->num_cities, &state);
    double temp = 1000.0;
    double alpha = 0.995;
    double current_cost = total_distance(data, current_tour);
    

    // Copiar a mejor tour inicial
    for(int i = 0; i < data->num_cities; i++) 
    {
        best_tours[idHilo * data->num_cities + i] = current_tour[i]; // en vez de best_tour[i], se calcula el indice a partir del id del hilo
    }
    best_costs[idHilo] = current_cost; // best cost se usa para mandar el toru con menor costo despues al host
    

    // Algoritmo de simulated annealing
    for(int iter = 0; iter < max_iter; iter++) 
    {
        // Generar indices aleatorios i y j usando curand
        int i = hiprand(&state) % data->num_cities;
        int j = hiprand(&state) % data->num_cities;
        swap(&current_tour[i], &current_tour[j]);
        
        double new_cost = total_distance(data, current_tour);
        
        // Conocer cual es el de mejor costo a partir de los costos almacenados en el arreglo bes_cost
        if(new_cost < current_cost || exp((current_cost - new_cost) / temp) > hiprand_uniform(&state))
        {
            current_cost = new_cost;
            for(int k = 0; k < data->num_cities; k++) 
            {
                best_tours[idHilo * data->num_cities + k] = current_tour[k]; // en vez de best_tour[k], se calcula el indice a partir del id del hilo
            }
            best_costs[idHilo] = current_cost;
        } 
        else 
        {
            swap(&current_tour[i], &current_tour[j]); // revert
        }
        temp *= alpha;
    }
}



void simulated_annealing(TSPData *data, int *best_tour) 
{
    // Calcular el numero de hilos a ocupar
    hipDeviceProp_t propiedades; // sacar propiedades del dispositivo
    hipGetDeviceProperties (&propiedades,0);
    int num_threads= propiedades.maxThreadsPerBlock; // calcular tamaño optimo del bloque
    int num_blocks= (MAX_CITIES+num_threads-1) / num_threads; // calcular el numero de bloques ; N es el numero de datos (en este caso el tamaño de las ciudades), la formula es universal
    // Si se quiere saber el numero de hilos maximo se multiplica tam_bloque*numero_bloques

    
    // Reservar memoria en el dispositivo
    TSPData *d_data;
    int *d_best_tours;
    double *d_best_costs;
    int max_iter = 100000;
    
    hipMalloc(&d_data, sizeof(TSPData));
    hipMalloc(&d_best_tours, num_threads * data->num_cities * sizeof(int));
    hipMalloc(&d_best_costs, num_threads * sizeof(double));
    
    // Copiar datos del host al dispositivo
    hipMemcpy(d_data, data, sizeof(TSPData), hipMemcpyHostToDevice);
    
    // Lanzar kernel
    unsigned long seed= time(NULL); // semilla utilizando la hora para utilizar con curand 
    kernel_simulated_annealing<<<num_blocks, num_threads>>>(d_data, d_best_tours, d_best_costs, max_iter, seed);
    
    // Copiar datos del dispositivo al host
    int *host_best_tours = (int*)malloc(num_threads * data->num_cities * sizeof(int));
    double *host_best_costs = (double*)malloc(num_threads * sizeof(double));
    hipMemcpy(host_best_tours, d_best_tours, num_threads * data->num_cities * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_best_costs, d_best_costs, num_threads * sizeof(double), hipMemcpyDeviceToHost);
    

    // Encontrar el mejor tour entre todos los hilos (ya en el host)
    double min_cost= host_best_costs[0]; // se dira que el primer valor es el mejor (de inicio)
    int best_thread= 0; // se dira que el hilo que tuvo el mejor tour sera el 0 (de inicio)
    for(int i=1; i<num_threads; i++) 
    {
        if(host_best_costs[i] < min_cost) // si el elemento actual es menor al de menor costo (previo)
        {
            min_cost= host_best_costs[i];
            best_thread= i; // se obtiene que hilo fue el de menor costo para poder imprimir su tour despues
        }
    }
    // Copiar el mejor tour al resultado
    for(int i=0; i<data->num_cities; i++) 
    {
        best_tour[i]= host_best_tours[best_thread * data->num_cities + i]; // de todos los tours se obtiene el mejor dependiendo de que hilo fue 
    }
    
    printf("Costo final: %.2f\n", min_cost);

    // Liberar memoria 
    // Host
    free(host_best_tours);
    free(host_best_costs);
    // Dispositivo
    hipFree(d_data);
    hipFree(d_best_tours);
    hipFree(d_best_costs);
}

int main() 
{
    srand(time(NULL));
    TSPData data;
    FILE *fp = fopen("berlin52.tsp", "r");
    if(!fp) {
        printf("No se pudo abrir el archivo.\n");
        return 1;
    }

    char line[128];
    while(fgets(line, sizeof(line), fp)) {
        if(strncmp(line, "NODE_COORD_SECTION", 18) == 0)
            break;
    }

    int index;
    double x, y;
    data.num_cities = 0;
    while(fscanf(fp, "%d %lf %lf", &index, &x, &y) == 3) {
        data.x[data.num_cities] = x;
        data.y[data.num_cities] = y;
        data.num_cities++;
    }
    fclose(fp);

    int best_tour[MAX_CITIES];
    simulated_annealing(&data, best_tour);

    printf("Tour final:\n");
    for (int i = 0; i < data.num_cities; i++)
        printf("%d->", best_tour[i] + 1); //+1 para coincidir con los índices de TSPLIB
    
    printf("%d\n", best_tour[0] + 1); //regresamos al inicio

    return 0;
}