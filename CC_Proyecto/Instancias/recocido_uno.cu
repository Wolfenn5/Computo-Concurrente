#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#define MAX_CITIES 1000

typedef struct {
    double x[MAX_CITIES];
    double y[MAX_CITIES];
    int num_cities;
} TSPData;

__device__ double distance(double x1, double y1, double x2, double y2){
    return sqrt((x1 - x2)*(x1 - x2) + (y1 - y2)*(y1 - y2));
}

__device__ double total_distance(TSPData *data, int *tour){
    double dist = 0.0;
    for(int i = 0; i < data->num_cities - 1; i++){
        dist += distance(data->x[tour[i]], data->y[tour[i]], data->x[tour[i+1]], data->y[tour[i+1]]);
    }
    dist += distance(data->x[tour[data->num_cities-1]], data->y[tour[data->num_cities-1]], data->x[tour[0]], data->y[tour[0]]);
    return dist;
}

__device__ void swap(int *a, int *b){
    int tmp = *a;
    *a = *b;
    *b = tmp;
}

__device__ void shuffle(int *tour, int n){
    for(int i = 0; i < n; i++) tour[i] = i;
    for(int i = 0; i < n; i++){
        int j = rand() % n;
        swap(&tour[i], &tour[j]);
    }
}


__global__ void simulated_annealing_kernel(TSPData *data, int *best_tours, double *best_costs, int max_iter) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Cada hilo tendrá su propio tour y estado
    int current_tour[MAX_CITIES];
    double temp = 1000.0;
    double alpha = 0.995;
    
    // Inicializar tour
    for(int i = 0; i < data->num_cities; i++) current_tour[i] = i;
    
    // Funcion shuffle, se quito porque shuffle llama a swap y llamar de device entre device no se puede
    // for(int i = 0; i < n; i++) tour[i] = i;
    // for(int i = 0; i < n; i++){
    //     int j = rand() % n;
    //     swap(&tour[i], &tour[j]);
    // }

    for(int i = 0; i < data->num_cities; i++) {
        int j = (tid * 100 + i) % data->num_cities; // Pseudo-aleatorio basado en tid que es el id del hilo
        swap(&current_tour[i], &current_tour[j]);
    }
    


    
    double current_cost = total_distance(data, current_tour);
    
    // Copiar a mejor tour inicial
    for(int i = 0; i < data->num_cities; i++) {
        best_tours[tid * data->num_cities + i] = current_tour[i];
    }
    best_costs[tid] = current_cost;
    
    // Algoritmo de simulated annealing
    for(int iter = 0; iter < max_iter; iter++) {
        int i = (tid * 100 + iter) % data->num_cities; // Pseudo-aleatorio
        int j = (tid * 200 + iter) % data->num_cities; // Pseudo-aleatorio
        swap(&current_tour[i], &current_tour[j]);
        
        double new_cost = total_distance(data, current_tour);
        
        // Criterio de aceptación
        if(new_cost < current_cost || 
           exp((current_cost - new_cost) / temp) > (double)((tid + iter) % 1000) / 1000.0) {
            current_cost = new_cost;
            // Actualizar mejor tour
            for(int k = 0; k < data->num_cities; k++) {
                best_tours[tid * data->num_cities + k] = current_tour[k];
            }
            best_costs[tid] = current_cost;
        } else {
            swap(&current_tour[i], &current_tour[j]); // revertir
        }
        
        temp *= alpha;
    }
}

void simulated_annealing(TSPData *data, int *best_tour) {
    // Calcular el numero de hilos a ocupar
    // Sacar propiedades del dispositivo
    hipDeviceProp_t propiedades;
    hipGetDeviceProperties (&propiedades,0);
    int num_threads= propiedades.maxThreadsPerBlock; // calcular tamaño optimo del bloque
    // int tam_bloque= 1024; // para el ejercicio de medir tiempos de ejecucion
    int num_blocks= (MAX_CITIES+num_threads-1) / num_threads; // calcular el numero de bloques ; N es el numero de datos (en este caso el tamaño de la matriz), la formula es universal
    // Si se quiere saber el numero de hilos maximo se multiplica tam_bloque*numero_bloques






    
    // Reservar memoria en el dispositivo
    TSPData *d_data;
    int *d_best_tours;
    double *d_best_costs;
    
    hipMalloc(&d_data, sizeof(TSPData));
    hipMalloc(&d_best_tours, num_threads * data->num_cities * sizeof(int));
    hipMalloc(&d_best_costs, num_threads * sizeof(double));
    
    // Copiar datos al dispositivo
    hipMemcpy(d_data, data, sizeof(TSPData), hipMemcpyHostToDevice);
    
    // Lanzar kernel
    simulated_annealing_kernel<<<num_blocks, num_threads>>>(d_data, d_best_tours, d_best_costs, 100000);
    
    // Copiar resultados de vuelta al host
    int *host_best_tours = (int*)malloc(num_threads * data->num_cities * sizeof(int));
    double *host_best_costs = (double*)malloc(num_threads * sizeof(double));
    
    hipMemcpy(host_best_tours, d_best_tours, num_threads * data->num_cities * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_best_costs, d_best_costs, num_threads * sizeof(double), hipMemcpyDeviceToHost);
    
    // Encontrar el mejor tour entre todos los hilos
    double min_cost = host_best_costs[0];
    int best_thread = 0;
    for(int i = 1; i < num_threads; i++) {
        if(host_best_costs[i] < min_cost) {
            min_cost = host_best_costs[i];
            best_thread = i;
        }
    }
    
    // Copiar el mejor tour al resultado
    for(int i = 0; i < data->num_cities; i++) {
        best_tour[i] = host_best_tours[best_thread * data->num_cities + i];
    }
    
    printf("Costo final: %.2f\n", min_cost);
    
    // Liberar memoria
    free(host_best_tours);
    free(host_best_costs);
    hipFree(d_data);
    hipFree(d_best_tours);
    hipFree(d_best_costs);
}

int main() {
    srand(time(NULL));
    TSPData data;
    FILE *fp = fopen("berlin52.tsp", "r");
    if(!fp) {
        printf("No se pudo abrir el archivo.\n");
        return 1;
    }

    char line[128];
    while(fgets(line, sizeof(line), fp)) {
        if(strncmp(line, "NODE_COORD_SECTION", 18) == 0)
            break;
    }

    int index;
    double x, y;
    data.num_cities = 0;
    while(fscanf(fp, "%d %lf %lf", &index, &x, &y) == 3) {
        data.x[data.num_cities] = x;
        data.y[data.num_cities] = y;
        data.num_cities++;
    }
    fclose(fp);

    int best_tour[MAX_CITIES];
    simulated_annealing(&data, best_tour);

    printf("Tour final:\n");
    for (int i = 0; i < data.num_cities; i++)
        printf("%d->", best_tour[i] + 1); //+1 para coincidir con los índices de TSPLIB
    
    printf("%d\n", best_tour[0] + 1); //regresamos al inicio

    return 0;
}