#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

/* La directiva __host__ sirve para ejecutar funciones en la cpu, en si no tiene mucho sentido su uso ya que seria equivalente a decid void imprime() */



__device__ void imprimeVectorDevice (int *vector)
{
    printf("%d "),vector[blockIdx.x * blockDim.x + threadIdx.x];
}




__host__ void imprimeVectorHost (int *vector, int n)
{
    for (int i=0; i<n; i++)
    {
        printf("%d ",vector[i]);
    }
    printf("\n");
    
}



__global__ void sumaVectores (int*a, int*b, int*c , int n)
{
    // Se hace asi porque no se trabaja con un unico bloque, se tiene mas de un bloque
    int id_hilo= blockIdx.x * blockDim.x + threadIdx.x;

    if (id_hilo<n)
    {
        c[id_hilo]= a[id_hilo]+b[id_hilo];
        // Aqui se hara la prueba de ejecutar imprimeVector_Host desde la GPU, no deja al momento de compilar
        //imprimeVectorHost(c,n);
        imprimeVectorDevice(c);
    }
}


int main(int argc, char const *argv[])
{
    srand(time(NULL));

    // Parametro del main para el tamaño de los arreglos
    int n= atoi(argv[1]);

    // Arreglos dispositivo y host
    int *a_dispositivo, *b_dispositivo, *c_dispositivo;
    int *a_host= (int *)malloc(sizeof(int)*n);
    int *b_host= (int *)malloc(sizeof(int)*n);
    int *c_host= (int *)malloc(sizeof(int)*n);

    // Inicialziar arreglos
    for (int i=0; i<n; i++)
    {
        a_host[i]= 1+rand() % n;
        b_host[i]= 1+rand() % n;
        c_host[i]= 1+rand() % n;
    }
    

    // Declarar en GPU
    hipMalloc(&a_dispositivo, n*sizeof(int));
    hipMalloc(&b_dispositivo, n*sizeof(int));
    hipMalloc(&c_dispositivo, n*sizeof(int));

    // Copiar del host al dispositivo
    hipMemcpy(a_dispositivo, a_host, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_dispositivo, b_host, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_dispositivo, c_host, n*sizeof(int), hipMemcpyHostToDevice);


    // Sacar las propiedades del dispositivo
    hipDeviceProp_t propiedades; 
    hipGetDeviceProperties (&propiedades,0);

    int tamanio_bloque= propiedades.maxThreadsPerBlock;
    int numero_bloques= (n*tamanio_bloque-1) / tamanio_bloque;


    sumaVectores<<<numero_bloques, tamanio_bloque>>>(a_dispositivo, b_dispositivo, c_dispositivo, n);
    hipDeviceSynchronize();

    // Copiar del dispositivo al host
    hipMemcpy(a_host, a_dispositivo, n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(b_host, b_dispositivo, n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(c_host, c_dispositivo, n*sizeof(int), hipMemcpyDeviceToHost);


    printf("\nLos resultados del host: ");
    imprimeVectorHost(c_host,n);

    // Liberar recursos del dispositivo
    hipFree(a_dispositivo);
    hipFree(b_dispositivo);
    hipFree(c_dispositivo);


    return 0;
}

